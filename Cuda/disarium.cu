
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <cstddef>
#include <cmath>

#define VECLENGTH 100000

__global__ void kernel(long *disarium, long *result); 
void fillVector(long *disarium);


int main(int argc, char** argv) {
  long *disarium;
  long *result;
  disarium = (long*)malloc(VECLENGTH*sizeof(long));
  result = (long*)malloc(VECLENGTH*sizeof(long));
  fillVector(disarium);
  int threads;
  if(argc > 1)
    threads = atoi(argv[1]);
  else
    threads = 1;
  
  long *dev_disarium;
  long *dev_result;
  hipMalloc(&dev_disarium, VECLENGTH*sizeof(long));
  hipMalloc(&dev_result, VECLENGTH*sizeof(long));
  
  hipMemcpy(dev_disarium, disarium, VECLENGTH*sizeof(long), hipMemcpyHostToDevice);
  
  int blocksize = ceil(VECLENGTH/threads);

  hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
  
  kernel<<<blocksize, threads>>>(dev_disarium, dev_result);
  
  hipEventRecord(stop, 0);
	hipEventSynchronize(stop); //asynchroniczność CPU,GPU

  hipMemcpy(result, dev_result, VECLENGTH*sizeof(long), hipMemcpyDeviceToHost);
  
  hipFree(dev_disarium);
  hipFree(dev_result);

  FILE *file = fopen("out", "w");

  for(int i=0;i<VECLENGTH;i++) {
    char line[50];
    sprintf(line, "%d - %d \n", *(disarium+i), *(result+i));
    fputs(line, file);
  }
  
  fclose(file);
  free(disarium);
  free(result);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("%f\n", milliseconds);

  return 0;
}
__global__ void kernel(long *disarium, long *result) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(tid < VECLENGTH)
  {
    int number = *(disarium+tid);
    long last = 0, sum = 0;
    int size = floor(__log10f(abs(number))) + 1;
    long tempNumber = number;
    int i = 0;
    while(tempNumber > 0) {
      last = tempNumber % 10;
      tempNumber = tempNumber / 10;
      sum += pow(last, size - i);
      i++;
    }
    if(sum == number) {
      *(result+tid) = 1;
    }
    else
      *(result+tid) = 0;
  }
}

void fillVector(long *disarium){
  
  for(int i=0;i<VECLENGTH;i++){
    long number = i+1;
    disarium[i] = number;
  }
  //printf("%f\n",end-start); 
}
